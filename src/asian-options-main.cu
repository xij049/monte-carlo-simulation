#include "hip/hip_runtime.h"
/* Headers and definitions */
#include "asian-options.h"

/* RNG initializer */
__global__ void setup_rng(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

/* Main procedure */
int main(const int argc, const char** argv) {
  /* Final paths */
  double finalPaths[NUM_PATHS * NUM_STEPS];
  const size_t pathsSize = NUM_PATHS * NUM_STEPS;

  /* RNG states */
  hiprandState *devStates;
  errorCheck(hipMalloc(&devStates, NUM_PATHS * sizeof(hiprandState)));
  setup_rng<<< 1, NUM_PATHS >>>(devStates, time(NULL));
  errorCheck(hipPeekAtLastError());

  /* Device memory:
   *   Old paths
   *   New paths
   *   Old Z's
   *   New Z's
   */
  double *paths;
  errorCheck(hipMalloc(&paths, 4 * pathsSize * sizeof(double)));

  /* Generate and print the paths */
  for (unsigned long i = 0; i < NUM_BLOCKS; ++i) {
    MCMC_options<<< 1, NUM_PATHS >>>(paths, paths + pathsSize, paths + 2 * pathsSize, paths + 3 * pathsSize, devStates);
    errorCheck(hipPeekAtLastError());
    errorCheck(hipDeviceSynchronize());

    /* Copy the paths from CUDA to host */
    errorCheck(hipMemcpy(finalPaths, &paths[pathsSize], pathsSize * sizeof(double), hipMemcpyDeviceToHost));
    errorCheck(hipDeviceSynchronize());

    /* Print the paths */
    for (unsigned long j = 0; j < NUM_PATHS; ++j) {
      for (unsigned long k = 0; k < (NUM_STEPS - 1); ++k) {
        printf("%.4lf,", finalPaths[NUM_STEPS * j + k]);
      }
      printf("%.4lf\n", finalPaths[NUM_STEPS * (j + 1) - 1]);
    }
  }

  return 0;
}
